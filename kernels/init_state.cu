void init_state_launcher() {
	CHECK_INIT

	hipError_t err = hipMemset(board, 0, sizeof(board[0])*BATCH_MAP_SZ); CHECK_CUDA_ERR
	err = hipMemset(board_prev, 0, sizeof(board[0])*BATCH_MAP_SZ); CHECK_CUDA_ERR
	err = hipMemset(board_pprev, 0, sizeof(board[0])*BATCH_MAP_SZ); CHECK_CUDA_ERR
	
	err = hipMemset(n_captures, 0, sizeof(n_captures[0])*N_PLAYERS*BATCH_SZ); CHECK_CUDA_ERR
}

